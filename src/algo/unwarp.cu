#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <array.h>
#include "config.h"
//#include "tictoc.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#define BLOCK_SIZE (128)
#define MAX_DEVICES (16)  //increase this if you've got more devices.

///// Imports from unwarp.c
extern "C" int  compute_map (float *xs, int w, float duty);
extern "C" void compute_norm(float *norm, float *xs, int w);

///// Error handling
#define REPORT_ERR(exprstr,ecode) \
  { if(ecode!=hipSuccess)                       \
    { fprintf(stderr,                            \
        "**ERROR [CUDA Unwarp]"ENDL                     \
        "- %s(%d)"ENDL                           \
        "- Expression: %s"ENDL                   \
        "- \t%s "ENDL,                           \
        __FILE__,__LINE__,exprstr,hipGetErrorString(e));\
      goto Error;                                \
    }                                            \
  }

#define REPORT_WRN(exprstr,ecode) \
  { if(ecode!=hipSuccess)                       \
    { fprintf(stderr,                            \
        "**WARNING [CUDA Unwarp]"ENDL                   \
        "- %s(%d)"ENDL                           \
        "- Expression: %s"ENDL                   \
        "- \t%s "ENDL,                           \
        __FILE__,__LINE__,exprstr,hipGetErrorString(e));\
    }                                            \
  }

#define ASRT(expr) \
  if(!(expr))                                    \
  { fprintf(stderr,                              \
      "**ERROR [Unwarp]\tAssertion failed"ENDL   \
      "- %s(%d): %s"ENDL                         \
      ,__FILE__,__LINE__,#expr);                 \
    goto Error;                                  \
  }


#define CHECK(expr) \
  { hipError_t e = (expr);                      \
    REPORT_ERR(#expr,e);                         \
  }

#define WARN(expr) \
  { hipError_t e = (expr);                      \
    REPORT_WRN(#expr,e);                         \
  }

#define EXPECT(expr,expect,lbl) \
{ hipError_t e = (expr);         \
  if(e!=hipSuccess)            \
  { if(e==expect) goto lbl;     \
    REPORT_ERR(#expr,e);        \
  }                             \
}

///// Utils
//static
//size_t bytesof(const Array *a)
//{ static const int Bpp[] = {1,2,3,4,1,2,3,4,4,8};
//  return a->size*Bpp[a->type];
//}
static
size_t bytesof_row(const Array *a)
{ static const int Bpp[] = {1,2,3,4,1,2,3,4,4,8};
  return a->dims[0]*Bpp[a->type];
}

///// row-wise lut kernel
// - Use 1D thread indexes.  Each addresses a row in the input data.
// - Each thread remaps the input data in a row to output pixels with averaging.
typedef unsigned int uint;

__device__ uint nearest(float x) { return rintf(x); }

// Lookup is by Nearest
// Composition is additive
template<typename Tin,typename Tout>
__global__ void rowwise_lut_kernel(
  const Tin*    __restrict__  in, 
        Tout*   __restrict__  out,
  const uint                  width_in,
  const uint                  width_out,
  const float* __restrict__   lut,
  const float* __restrict__   norm,
  const uint                  nrows)
{   
  const uint  irow = threadIdx.x + blockIdx.x*blockDim.x;
  if(irow>=nrows) return; // bounds check

  const Tin*   rin = in + width_in * irow;
        Tout* rout = out+ width_out* irow;

  for(uint i=0;i<width_in;++i)
  { uint j = nearest(lut[i]);
    rout[j] += rin[i]/norm[j];
  }
}

#define KERNELCALL(t1,t2) rowwise_lut_kernel<t1,t2><<< grid,threads,0,streams[i] >>>((t1*)dev_ins[i],(t2*)dev_outs[i],width_in,width_out,dev_luts[i],dev_norms[i],nrows)
#define KERNELCALLS(t1) \
  switch(out->type)                                         \
  {                                                         \
    case UINT8_TYPE:    KERNELCALL(t1,uint8 ); break;       \
    case UINT16_TYPE:   KERNELCALL(t1,uint16); break;       \
    case UINT32_TYPE:   KERNELCALL(t1,uint32); break;       \
    case UINT64_TYPE:   KERNELCALL(t1,uint64); break;       \
    case  INT8_TYPE:    KERNELCALL(t1, int8 ); break;       \
    case  INT16_TYPE:   KERNELCALL(t1, int16); break;       \
    case  INT32_TYPE:   KERNELCALL(t1, int32); break;       \
    case  INT64_TYPE:   KERNELCALL(t1, int64); break;       \
    case  FLOAT32_TYPE: KERNELCALL(t1, float); break;       \
    case  FLOAT64_TYPE: KERNELCALL(t1, double); break;      \
    default: goto Error;                                    \
  }

extern "C" int unwarp_gpu(Array *out, Array *in, float duty)
{   
  hipStream_t streams[MAX_DEVICES];
  void  *dev_ins[MAX_DEVICES],  *dev_outs[MAX_DEVICES];
  float *dev_luts[MAX_DEVICES], *dev_norms[MAX_DEVICES];
  float *xs,*norm;
  const int w = in->dims[0];
  const size_t bytesof_lut = sizeof(float)*w;

  // Compute the LUT
  ASRT( xs=(float*)malloc(2*bytesof_lut) ); // alloc for xs and norm
  norm = xs + w;
  compute_map(xs,w,duty);
  compute_norm(norm,xs,w);


  // feed the devices
  { int totalrows = in->dims[1]*in->dims[2];
    int ndevices;
    CHECK( hipGetDeviceCount(&ndevices) );
    ASRT( ndevices < MAX_DEVICES );  
    // init streams
    for(int i=0;i<ndevices;++i)
    {
      hipSetDevice(i);
      CHECK( hipStreamCreate(streams+i));
    }
    // alloc for devices
    for(int i=0;i<ndevices;++i)
    { hipSetDevice(i);      
      uint nrows = totalrows/ndevices,
           irow  = nrows*i;
      if(i==ndevices-1)           // take care of any rounding problems on the last iteration
        nrows = totalrows - irow;    

      CHECK( hipMalloc(&dev_ins[i]  ,nrows*bytesof_row(in)) );
      CHECK( hipMalloc(&dev_outs[i] ,nrows*bytesof_row(out)) );
      CHECK( hipMalloc(&dev_luts[i] ,bytesof_lut) );
      CHECK( hipMalloc(&dev_norms[i],bytesof_lut) );
      CHECK( hipMemset(dev_outs[i],0,nrows*bytesof_row(out)) );
    }
    // upload data
    for(int i=0;i<ndevices;++i)
    { hipSetDevice(i);      
      uint nrows = totalrows/ndevices,
           irow  = nrows*i;
      if(i==ndevices-1)           // take care of any rounding problems on the last iteration
        nrows = totalrows - irow;
      CHECK( hipMemcpyAsync(dev_ins[i],  
                             AUINT8(in) + irow*bytesof_row(in), 
                             nrows*bytesof_row(in),
                             hipMemcpyHostToDevice,
                             streams[i]));  
      CHECK( hipMemcpyAsync(dev_luts[i], xs,   bytesof_lut,hipMemcpyHostToDevice,streams[i]) );  
      CHECK( hipMemcpyAsync(dev_norms[i],norm, bytesof_lut,hipMemcpyHostToDevice,streams[i]) );  
    }
    // execute
    for(int i=0;i<ndevices;++i)
    { hipSetDevice(i);      
      uint nrows = totalrows/ndevices,
           irow  = nrows*i;
      if(i==ndevices-1)           // take care of any rounding problems on the last iteration
        nrows = totalrows - irow;
    
      uint width_in  =  in->dims[0],
           width_out = out->dims[0];
      dim3 threads(BLOCK_SIZE,1,1),
           grid((unsigned int)ceil(nrows/(float)BLOCK_SIZE),1,1);

      switch(in->type)
      {                                                     
        case   UINT8_TYPE: KERNELCALLS(uint8 ) break;   
        case  UINT16_TYPE: KERNELCALLS(uint16) break;   
        case  UINT32_TYPE: KERNELCALLS(uint32) break;   
        case  UINT64_TYPE: KERNELCALLS(uint64) break;   
        case    INT8_TYPE: KERNELCALLS( int8 ) break;   
        case   INT16_TYPE: KERNELCALLS( int16) break;   
        case   INT32_TYPE: KERNELCALLS( int32) break;   
        case   INT64_TYPE: KERNELCALLS( int64) break;   
        case FLOAT32_TYPE: KERNELCALLS( float) break; 
        case FLOAT64_TYPE: KERNELCALLS(double) break;
        default:
          goto Error;
      }
    }
    // download results
    for(int i=0;i<ndevices;++i)  
    { hipSetDevice(i);      
      uint nrows = totalrows/ndevices,
           irow  = nrows*i;
      if(i==ndevices-1)           // take care of any rounding problems on the last iteration
        nrows = totalrows - irow;
      CHECK(hipMemcpyAsync(AUINT8(out) + irow*bytesof_row(out),
                            dev_outs[i],
                            nrows*bytesof_row(out),
                            hipMemcpyDeviceToHost,
                            streams[i]));
    }
    // cleanup
    for(int i=0;i<ndevices;++i)
    { hipSetDevice(i);
      CHECK(hipStreamSynchronize(streams[i]));
      CHECK( hipStreamDestroy(streams[i]) );
      CHECK( hipFree(dev_ins[i]) );
      CHECK( hipFree(dev_outs[i]) );
      CHECK( hipFree(dev_luts[i]) );
      CHECK( hipFree(dev_norms[i]) );
    } 
  }
    
  free(xs);
  
  return 1;
Error:
  return 0;
}
