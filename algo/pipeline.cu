#include "hip/hip_runtime.h"
#include "pipeline.h"
#include "pipeline-image.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>  //for printf
#include <stdlib.h> //for malloc
#include <string.h> //for memset
#include <stdint.h>

#include "hip/hip_runtime.h"

#define BX_   (32)
#define BY_   (4)
#define WORK_ (8)

#if 0
#define ECHO(estr)   LOG("---\t%s\n",estr)
#else
#define ECHO(estr)
#endif
static void breakme() {}

#define LOG(...)     printf(__VA_ARGS__)
#define REPORT(estr,msg) LOG("%s(%d): %s()\n\t%s\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,estr,msg)
#define TRY(e)       do{ECHO(#e);if(!(e)){REPORT(#e,"Evaluated to false.");breakme(); goto Error;}}while(0)
#define FAIL(msg)    do{REPORT("Failure.",msg);goto Error;} while(0)
#define NEW(T,e,N)   TRY((e)=(T*)malloc(sizeof(T)*(N)))
#define ZERO(T,e,N)  memset((e),0,sizeof(T)*(N))

#define CUREPORT(ecode,estr) LOG("%s(%d): %s()\n\t%s\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,estr,hipGetErrorString(ecode))
#define CUTRY(e)             do{ hipError_t ecode; ECHO(#e); ecode=(e); if(ecode!=hipSuccess){CUREPORT(ecode,#e);goto Error;}}while(0)
#define CUWARN(e)            do{ hipError_t ecode; ECHO(#e); ecode=(e); if(ecode!=hipSuccess){CUREPORT(ecode,#e);           }}while(0)
#define CUNEW(T,e,N)    CUTRY(hipMalloc((void**)&(e),sizeof(T)*(N)))
#define CUZERO(T,e,N)   CUTRY(hipMemset((e),0,sizeof(T)*(N)))

#define countof(e)   (sizeof(e)/sizeof(*(e)))

#define CEIL(num,den) (((num)+(den)-(1))/(den))
/**
 * The parameter collection that gets passed to the kernel
 */
struct pipeline_ctx_t
{ unsigned * __restrict__ ilut;        ///< look up table for unwarp (ctx.w/2 number of elements).  Set by launch on first call or if width changes.
  float    * __restrict__ lut_norms0,
           * __restrict__ lut_norms1;
  unsigned istride,     ///< number of elements between rows of source.
           ostride;     ///< number of elements between rows of output.
  unsigned w,h;         ///< source width and height (height is nrows*nchan)
};

/**
 * The object that manages pipeline execution.
 */
typedef struct pipeline_t_
{ pipeline_ctx_t ctx;
  unsigned       count, ///< the number of frames that have been pushed to the accumulator
                 every; ///< the number of frames to average
  double         samples_per_scan;
  bool           invert;
  unsigned       downsample;
  unsigned       alignment;         ///< output rows are aligned to this target number of elements.
  unsigned       nbytes_tmp;
  float    norm,        ///< 1.0/the frame count as a float - set by launcher (eg. for ctx.every=4, this should be 0.25)
           m,b;         ///< slope and intercept for intensity scaling
  void  * __restrict__ src,         ///< device buffer
        * __restrict__ dst;         ///< device buffer
  float * __restrict__ tmp;         ///< device buffer
} *pipeline_t;

//
// --- KERNELS ---
//

// should schedule for destination (2*width/WORK/BX,height/2/BY) blocks [eg for 4864x512 -> (38,8)]
// dst width must be aligned  to WORK*BX   (256)
//     height must be aligned to BY*2      (64)
// ilut must be aligned to WORK*BX and sized 2*dst width
//
template<typename T,    ///< pixel type (input and output)
         unsigned BX,   ///< block size in X
         unsigned BY,   ///< block size in Y (channel dimension unrolled into Y)
         unsigned WORK  ///< number of elements to process per thread
         >
__global__ void __launch_bounds__(BX*BY,1) /* max threads, min blocks */
warp_kernel(pipeline_ctx_t ctx, const T* __restrict__ src, float* __restrict__ dst)
{ const unsigned ox=threadIdx.x+blockIdx.x*WORK*BX,
                 oy=threadIdx.y+blockIdx.y*BY;
  unsigned * __restrict__ ilut = ctx.ilut + ox;
  dst+=ox+oy*ctx.ostride*2;
  src+=   oy*ctx.istride;
  if(blockIdx.x<ctx.ostride/(WORK*BX)) // forward scan
  {
#pragma unroll
    for(int i=0;i<WORK;++i)
    { const int j0=ilut[i*BX],
                j1=ilut[i*BX+1];
      float v0=0.0f,v1=0.0f;
      if(j0>0) v1=ctx.lut_norms1[j0-1]*src[j0-1];
      for(int j=j0;j<j1;++j)
        v0+=ctx.lut_norms0[j]*src[j];
      dst[i*BX]+=v0+v1;
    }
  } else { // backward scan
#pragma unroll
    for(int i=0;i<WORK;++i)
    { const int j1=ilut[i*BX+1],
                j0=ilut[i*BX+2];
      float v0=0.0f;
      for(int j=j0;j<j1;++j)
        v0+=ctx.lut_norms0[j]*src[j];
      float v1=ctx.lut_norms1[j1]*src[j1];
      dst[i*BX]+=v0+v1;
    }
  }
}

/**
 * Cast array from float to T.
 * Rounds pixel values, so this isn't appropriate for converting to floating point types.
 * src and dst should be the same shape but may be different types.
 * Both must have width aligned to BX*WORK
 * Both must have heigh aligned to BY
 */
template<typename T,    ///< pixel type (input and output)
         unsigned BX,   ///< block size in X
         unsigned BY,   ///< block size in Y (channel dimension unrolled into Y)
         unsigned WORK  ///< number of elements to process per thread
         >
__global__ void __launch_bounds__(BX*BY,1)
cast_kernel(T*__restrict__ dst, const float* __restrict__ src, unsigned stride,const float m, const float b)
{ const int ox=threadIdx.x+blockIdx.x*WORK*BX,
            oy=threadIdx.y+blockIdx.y*BY;
  //if(oy>=h) return; // for unaligned y, uncomment and add an argument to the kernel call
  src+=ox+oy*stride;
  dst+=ox+oy*stride;
  #pragma unroll
  for(int i=0;i<WORK;++i)
    dst[i*BX]=round(fmaf(src[i*BX],m,b));
}

//
// --- PUBLIC INTERFACE ---
//

pipeline_t pipeline_make(const pipeline_param_t *params)
{ pipeline_t self=NULL;
  TRY(params);
  NEW(pipeline_t_,self,1);
  ZERO(pipeline_t_,self,1);
  self->every            = (params->frame_average_count<1)?1:params->frame_average_count;
  self->samples_per_scan = params->sample_rate_MHz*1.0e6/(double)params->scan_rate_Hz;
  self->invert           = (params->invert_intensity!=0);
  self->downsample       = (params->pixel_average_count<=1)?1:params->pixel_average_count;
  self->alignment        = BX_*WORK_;
  self->norm             = 1.0f/(float)self->every;
  self->m                = 1.0f;;
  self->b                = 0.0f;
  return self;
Error:
  return NULL;
}

void pipeline_free(pipeline_t *self)
{ if(self && *self)
  { void *ptrs[]={self[0]->ctx.ilut,
                  self[0]->ctx.lut_norms0,
                  self[0]->src,
                  self[0]->dst,
                  self[0]->tmp};
    for(int i=0;i<countof(ptrs);++i)
      if(ptrs[i])
        CUWARN(hipFree(ptrs[i]));
    free(*self); *self=NULL;
  }
}

#define EPS (1e-3)
static unsigned pipeline_get_output_width(pipeline_t self, const double inwidth)
{ const double d=1.0-inwidth/self->samples_per_scan; // 1 - duty
  //max derivative of the cosine warp adjusted to cos(2pi*(d/2)) is the zero point
  //and the positive part of the warp function goes from 0 to 1.
  const double maxslope=M_PI*(1.0-d)/inwidth/cos(M_PI*d);
  const double amplitude=1.0/maxslope;
  const unsigned w=self->alignment*(unsigned)(amplitude/self->downsample/self->alignment);
  TRY(-EPS<d && d<=(0.5+EPS));
  TRY(0<w && w<inwidth);
  return w;
Error:
  return 0;
}
#undef EPS

extern "C" int pipeline_get_output_dims(pipeline_t self, const pipeline_image_t src, unsigned *w, unsigned *h, unsigned *nchan)
{ TRY(self && src);
  if(nchan) *nchan=src->nchan;
  if(h)     *h=src->h*2;
  if(w)     TRY(*w=pipeline_get_output_width(self,src->w));
  return 1;
Error:
  return 0;
}

static int pipeline_alloc_lut(pipeline_t self, unsigned inwidth)
{ unsigned N=self->alignment*CEIL(inwidth,self->alignment); // pad to aligned width
  if(self->ctx.ilut)
  { CUTRY(hipFree(self->ctx.ilut));
    CUTRY(hipFree(self->ctx.lut_norms0));
  }
  const unsigned     ow = pipeline_get_output_width(self,inwidth);
  CUNEW(unsigned,self->ctx.ilut,      2*(ow+1));
  CUNEW(float   ,self->ctx.lut_norms0,2*N+1);
  return 1;
Error:
  self->ctx.ilut=NULL;
  self->ctx.lut_norms0=NULL;
  return 0;

}

static void dump(const char* name, void* data, size_t nbytes)
{ FILE* fp=0;
  TRY(fp=fopen(name,"wb"));
  fwrite(data,1,nbytes,fp);
  fclose(fp);
Error:;
}

static double f(double x) { return  0.5*(1.0-cos(2.0*M_PI*x)); }

static int pipeline_fill_lut(pipeline_t self, unsigned inwidth)
{ int isok=1;
  unsigned * __restrict__ lut=0;
  unsigned * __restrict__ ilut=0;
  float    * __restrict__ norms=0;
  // useful constants
  const double        d = (1.0-inwidth/self->samples_per_scan)/2.0; // 0.5*(1 - duty)
  const unsigned     ow = pipeline_get_output_width(self,inwidth);
  const double        s = (1.0-2.0*d)/(double)inwidth;
  const double        A = ow/(1.0-f(d));
  const double      Afd = A*f(d);
  const unsigned  halfw = inwidth/2;
  const unsigned      N = self->alignment*CEIL(inwidth,self->alignment); // pad to aligned width
  // alloc temporary space
  NEW(unsigned ,lut  ,inwidth);
  NEW(unsigned ,ilut ,2*(ow+1));
  NEW(float    ,norms,2*N+1);
  ZERO(unsigned,lut  ,inwidth);
  ZERO(unsigned,ilut ,2*(ow+1));
  ZERO(float   ,norms,2*N+1);

  // compute lookup
  for(unsigned i=0;i<inwidth;++i)
  { double p0=d+s*i,
           p1=d+s*(i+1);
    double v0=A*f(p0)-Afd,
           v1=A*f(p1)-Afd;

    int j,k;
    if(v0<0.0) v0=0.0;
    if(v1<0.0) v1=0.0;
    if(v0>v1) { double v=v0;v0=v1;v1=v; } //swap
    j = (int) v0;
    k = (int) v1;
    TRY( (k-j)<2 ); // longest length should be 1, so shouldn't straddle more than two pixels
    lut[i] = j + (i<halfw?0:ow);
    if( (k-j)==0 )
    { norms[i]   = v1-v0;
      norms[i+N] = 0.0;
    } else { //k-j==1 -> k=1+j
      norms[i]   = k-v0;
      norms[i+N] = v1-k;
    }
  }

  // interval encode lookup table on output side
  { unsigned last=0;
    for(unsigned i=0;i<halfw;++i)
      if(last!=lut[i])
        ilut[last=lut[i]]=i;
    ilut[ow  ]=inwidth/2; // add elements to deal with discontinuity
    ilut[ow+1]=inwidth; // subtract one to prevent reading off end
    ilut+=2;
    for(unsigned i=halfw;i<inwidth;++i)
      if(last!=lut[i])
        ilut[(last=lut[i])]=i;
    ilut-=2;
  }
#if 0
  dump("lut.u32",lut    ,inwidth*sizeof(*lut));
  dump("norms.f32",norms,2*N    *sizeof(*norms));
  dump("ilut.u32",ilut  ,2*(ow+1) *sizeof(*ilut));
#endif

  // upload
  CUTRY(hipMemcpy(self->ctx.ilut      ,ilut , 2*(ow+1)*sizeof(*ilut),hipMemcpyHostToDevice));
  CUTRY(hipMemcpy(self->ctx.lut_norms0,norms,(2*N+1)*sizeof(*norms) ,hipMemcpyHostToDevice));
  self->ctx.lut_norms1=self->ctx.lut_norms0+N;

Finalize:
  if(lut)   free(lut);
  if(ilut)  free(ilut);
  if(norms) free(norms);
  return isok;
Error:
  isok=0;
  goto Finalize;
}

static int pipeline_upload(pipeline_t self, pipeline_image_t dst, const pipeline_image_t src)
{ if(self->src && (self->ctx.w!=src->w || self->ctx.h!=src->h*src->nchan)) // if there's a shape change, realloc
  { CUTRY(hipFree(self->src)); self->src=0;
    CUTRY(hipFree(self->dst)); self->dst=0;
    CUTRY(hipFree(self->tmp)); self->tmp=0;
  }
  dst->h++; // pad by a line
  if(!self->src)
  { CUTRY(hipMalloc((void**)&self->src,pipeline_image_nbytes(src)+1024));
    CUTRY(hipMalloc((void**)&self->dst,pipeline_image_nbytes(dst)));
    CUTRY(hipMalloc((void**)&self->tmp,self->nbytes_tmp=pipeline_image_nelem(dst)*sizeof(float)));
    CUTRY(hipMemset(self->tmp,0,pipeline_image_nelem(dst)*sizeof(float)));
  }
  CUTRY(hipMemcpy(self->src,src->data,pipeline_image_nbytes(src),hipMemcpyHostToDevice));
  dst->h--; // restore original number of lines
  self->ctx.w=src->w;
  self->ctx.h=src->h*src->nchan;
  self->ctx.istride=src->stride;
  self->ctx.ostride=dst->stride;
  return 1;
Error:
  return 0;
}

static int pipeline_download(pipeline_t self, pipeline_image_t dst)
{ TRY(self->dst);
  CUTRY(hipMemcpy(dst->data,self->dst,pipeline_image_nbytes(dst),hipMemcpyDeviceToHost));
  return 1;
Error:
  return 0;
}


template<typename Tsrc, typename Tdst,unsigned BX,unsigned BY,unsigned WORK>
static int launch(pipeline_t self, int *emit)
{ unsigned ow=pipeline_get_output_width(self,self->ctx.w);
  dim3 threads(BX,BY),
       blocks(CEIL(2*ow,BX*WORK),CEIL(self->ctx.h,BY));      // for the cast from tmp to dst
  TRY(emit);
#if 1
  if(self->every>1) // frame averaging enabled
  { if( ((self->count+1)%self->every)==0 )
    { *emit=1;
      warp_kernel<Tsrc,BX,BY,WORK><<<blocks,threads>>>(self->ctx,(Tsrc*)self->src,self->tmp);
      cast_kernel<Tdst,BX,BY,WORK><<<blocks,threads>>>((Tdst*)self->dst,self->tmp,self->ctx.ostride*2,self->m*self->norm,self->b);
    } else
    { *emit=0;
      warp_kernel<Tsrc,BX,BY,WORK><<<blocks,threads>>>(self->ctx,(Tsrc*)self->src,self->tmp);
    }
    self->count++;
  } else            // frame averaging disabled
#endif
  { if(emit) *emit=1;
    warp_kernel<Tsrc,BX,BY,WORK><<<blocks,threads>>>(self->ctx,(Tsrc*)self->src,self->tmp);
    cast_kernel<Tdst,BX,BY,WORK><<<blocks,threads>>>((Tdst*)self->dst,self->tmp,self->ctx.ostride*2,self->m,self->b);
    CUTRY(hipGetLastError());
  }
  if(*emit)
    CUTRY(hipMemset(self->tmp,0,self->nbytes_tmp));
  return 1;
Error:
  return 0;
}

// generics

/** Requires a macro \c CASE(T) to be defined where \c T is a type parameter.
 *  Requires a macro \c FAIL to be defined that handles when an invalid \a type_id is used.
 *  \param[in] type_id Must be a valid nd_type_id_t.
 */
#define TYPECASE(type_id) \
switch(type_id) \
{            \
  case u8_id :CASE(uint8_t ); break; \
  case u16_id:CASE(uint16_t); break; \
  case u32_id:CASE(uint32_t); break; \
  case u64_id:CASE(uint64_t); break; \
  case i8_id :CASE(int8_t ); break; \
  case i16_id:CASE(int16_t); break; \
  case i32_id:CASE(int32_t); break; \
  case i64_id:CASE(int64_t); break; \
  case f32_id:CASE(float); break; \
  case f64_id:CASE(double); break; \
  default:   \
    FAIL("Unsupported pixel type.");    \
}
/** Requires a macro \c CASE2(T1,T2) to be defined where \c T1 and \c T2 are
 *  type parameters.
 *  Requires a macro \c FAIL to be defined that handles when an invalid \a type_id is used.
 *  \param[in] type_id Must be a valid nd_type_id_t.
 *  \param[in] T       A type name.  This should follow the u8,u16,u32,... form.  Usually
 *                     these types are defined in the implemenation function where this
 *                     macro is instanced.
 */
#define TYPECASE2(type_id,T) \
switch(type_id) \
{               \
  case u8_id :CASE2(T,uint8_t); break;  \
  case u16_id:CASE2(T,uint16_t); break; \
  case u32_id:CASE2(T,uint32_t); break; \
  case u64_id:CASE2(T,uint64_t); break; \
  case i8_id :CASE2(T,int8_t); break;  \
  case i16_id:CASE2(T,int16_t); break; \
  case i32_id:CASE2(T,int32_t); break; \
  case i64_id:CASE2(T,int64_t); break; \
  case f32_id:CASE2(T,float); break; \
  case f64_id:CASE2(T,double); break; \
  default:      \
    FAIL("Unsupported pixel type.");       \
}

int isaligned(unsigned x, unsigned n) { return (x%n)==0; }
int pipeline_exec(pipeline_t self, pipeline_image_t dst, const pipeline_image_t src, int *emit)
{ TRY(emit);

  TRY(isaligned(src->w,BX_));
  TRY(isaligned(src->h,BY_));
  TRY(isaligned(dst->w,BX_*WORK_));
  TRY(dst->h==2*src->h);

  { int count=0;
    CUTRY(hipGetDeviceCount(&count));
    CUTRY(hipSetDevice(count-1));
  }

  if(src->w>self->ctx.w)
  { TRY(pipeline_alloc_lut(self,src->w));
    TRY(pipeline_fill_lut(self,src->w));
  }
  pipeline_image_conversion_params(dst,src,self->invert,&self->m,&self->b);
  TRY(pipeline_upload(self,dst,src)); // updates context size and stride as well
// launch kernel
  #define CASE2(TSRC,TDST) launch<TSRC,TDST,BX_,BY_,WORK_>(self,emit)
  #define CASE(T)          TYPECASE2(src->type,T)
    { TYPECASE(dst->type); }
  #undef CASE
  #undef CASE2
  if(*emit)
    TRY(pipeline_download(self,dst));
  return 1;
Error:
  return 0;
}

#undef TYPECASE
#undef TYPECASE2
